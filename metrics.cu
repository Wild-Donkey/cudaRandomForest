#include "hip/hip_runtime.h"
// #include <metrics.cuh>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

// __device__ entropy_kernel () {}

// float Entropy(int* x, int n) {

// }
// float InfGain(int** x, int *y, int n, int m, int idx);


__global__ void markClass(unsigned* x, unsigned* y, unsigned n, unsigned xtarget) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  y[idx] = (x[idx] == Index) ? 1 : 0;
}

__global__ void computeHistogram(float* data, unsigned int* hist, int dataSize, int numBins, float minVal, float maxVal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dataSize) {
        float val = data[idx];
        int bin = (int)((val - minVal) / (maxVal - minVal) * numBins);
        if (bin >= 0 && bin < numBins) {
            atomicAdd(&hist[bin], 1);
        }
    }
}

// CUDA核函数：计算熵贡献
__global__ void computeEntropyContributions(unsigned int* hist, float* contributions, int numBins, int dataSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numBins) {
        unsigned int count = hist[idx];
        if (count > 0) {
            float p = (float)count / dataSize;
            contributions[idx] = -p * log2f(p);
        } else {
            contributions[idx] = 0.0f;
        }
    }
}

// 计算数组的信息熵
float computeEntropy(float* x, int n, int class_count) {
    unsigned *d_x, *d_hist;
    float entropy = 0.0f;
    
    unsigned threadsPerBlock = 256;
    unsigned n_block = (n + threadsPerBlock - 1) / threadsPerBlock;
    unsigned c_block = (class_count + threadsPerBlock - 1) / threadsPerBlock;

    // 1. 分配设备内存
    hipMalloc((void**)&d_x, n_block * threadsPerBlock * sizeof(unsigned));
    hipMalloc((void**)&d_hist, c_block * threadsPerBlock * sizeof(unsigned));
    
    hipMemcpy(d_x, h_data, n_block * threadsPerBlock * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_hist, 0, c_block * threadsPerBlock * sizeof(unsigned int));

    
    if (numBins <= 3) {
      unsigned *d_mark;
      hipMalloc((void**)&d_mark, n * sizeof(unsigned));
      for (int i = 0; i < numBins; i++) {
        markClass<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_mark, n, i);
         = ;
        hipcub::DeviceReduce::Sum(d_hist + i, sizeof(unsigned), d_mark, &totalSum, n);
      }
      hipFree(d_mark);
    } else {
      computeHistogram<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_hist, dataSize, numBins, minVal, maxVal);
    }
    
    // 4. 计算数据最小最大值 (这里简化处理，实际应用中可能需要单独计算)
    float minVal = 0.0f; // 应根据实际数据调整
    float maxVal = 1.0f; // 应根据实际数据调整
    
    // 5. 计算直方图
    computeHistogram<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_hist, dataSize, numBins, minVal, maxVal);
    
    // 6. 计算每个bin的熵贡献
    computeEntropyContributions<<<(numBins + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock>>>
        (d_hist, d_contributions, numBins, dataSize);
    
    // 7. 拷贝贡献值回主机
    hipMemcpy(h_contributions, d_contributions, numBins * sizeof(float), hipMemcpyDeviceToHost);
    
    // 8. 计算总熵
    for (int i = 0; i < numBins; i++) {
        entropy += h_contributions[i];
    }
    
    // 9. 释放内存
    hipFree(d_data);
    hipFree(d_hist);
    hipFree(d_contributions);
    free(h_contributions);
    
    return entropy;
}

int main() {
    // 示例数据
    const int dataSize = 1000000;
    const int numBins = 256; // 直方图bin数量
    
    float *h_data = (float*)malloc(dataSize * sizeof(float));
    
    // 生成随机数据 (0-1之间)
    for (int i = 0; i < dataSize; i++) {
        h_data[i] = (float)rand() / RAND_MAX;
    }
    
    // 计算熵
    float entropy = computeEntropy(h_data, dataSize, numBins);
    printf("信息熵: %f\n", entropy);
    
    free(h_data);
    return 0;
}